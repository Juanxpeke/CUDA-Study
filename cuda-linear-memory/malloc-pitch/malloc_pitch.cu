// ================================================================
// Malloc Pitch method for allocating 1D memory used as a 2D matrix
// ================================================================


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Simple column sum kernel
__global__
void sumColumnsKernel(float* deviceMatrix, float* deviceSum, size_t pitch, int width, int height)
{
  // Calculate coordinates
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Define true width
  int trueWidth = pitch / sizeof(float);

  deviceSum[idx] = 0.0f;

  // Set values
  if (idx < width)
  {
    for (int i = 0; i < height; i++)
    {
      deviceSum[idx] += deviceMatrix[idx + i * trueWidth];
    }
  }
}

// Simple row sum kernel
__global__
void sumRowsKernel(float* deviceMatrix, float* deviceSum, size_t pitch, int width, int height)
{
  // Calculate coordinates
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Define true width
  int trueWidth = pitch / sizeof(float);

  deviceSum[idx] = 0.0f;

  // Set values
  if (idx < height)
  {
    for (int i = 0; i < width; i++)
    {
      deviceSum[idx] += deviceMatrix[idx * trueWidth + i];
    }
  }
}

// Matrix sum kernel, in this case, each thread traverses the entire matrix
__global__
void matrixSumKernel(float* deviceMatrix, double* deviceSum, size_t pitch, int width, int height)
{
  // Calculate coordinates
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Define true width
  int trueWidth = pitch / sizeof(float);

  deviceSum[idx] = 0.0f;

  for (int r = 0; r < height; ++r)
  {
    for (int c = 0; c < width; ++c)
    {
      deviceSum[idx] += deviceMatrix[r * trueWidth + c];
    }
  }
}

// Host code
int main()
{
  // Input matrix of 32771 x 32771 dimensions
  const int width = (1 << 14) + 3;
  const int height = (1 << 14) + 3;

  // Number of threads that will traverse entire matrix for special matrix sum kernel
  const int threadsMatrixSum = (1 << 6);

  // Blocks of size 16
  int blockSize = 16; 

  // Round up in case sizes are not multiple of blockSize
  int numBlocksColumns = (width + blockSize - 1) / blockSize;
  int numBlocksRows = (height + blockSize - 1) / blockSize;
  int numBlocksMatrix = (threadsMatrixSum + blockSize - 1) / blockSize;

  // Initialize data in host memory
  float* hostMatrix = new float[width * height];

  for (int i = 0; i < width * height; ++i)
  {
    hostMatrix[i] = 1.0f;
  }
  
  // Copy to device memory using pitched memory
  float* deviceMatrix;
  size_t pitch;
  hipMallocPitch(&deviceMatrix, &pitch, width * sizeof(float), height);
  hipMemcpy2D(deviceMatrix, pitch, hostMatrix, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);

  // Result device memory, for special sum matrix, you have to use double precision float
  float* deviceSumColumns;
  hipMalloc(&deviceSumColumns, width * sizeof(float));
  float* deviceSumRows;
  hipMalloc(&deviceSumRows, height * sizeof(float));
  double* deviceSumMatrix;
  hipMalloc(&deviceSumMatrix, threadsMatrixSum * sizeof(double));

  // Call columns kernel
  sumColumnsKernel<<<numBlocksColumns, blockSize>>>(deviceMatrix, deviceSumColumns, pitch, width, height);
  hipDeviceSynchronize();

  // Call rows kernel
  sumRowsKernel<<<numBlocksRows, blockSize>>>(deviceMatrix, deviceSumRows, pitch, width, height);
  hipDeviceSynchronize();

  // Call matrix special kernel
  matrixSumKernel<<<numBlocksMatrix, blockSize>>>(deviceMatrix, deviceSumMatrix, pitch, width, height);
  hipDeviceSynchronize();

  // Result host memory, for special sum matrix, you have to use double precision float
  float* hostSumColumns = new float[width];
  hipMemcpy(hostSumColumns, deviceSumColumns, width * sizeof(float), hipMemcpyDeviceToHost);
  float* hostSumRows = new float[height];
  hipMemcpy(hostSumRows, deviceSumRows, height * sizeof(float), hipMemcpyDeviceToHost);
  double* hostSumMatrix = new double[threadsMatrixSum];
  hipMemcpy(hostSumMatrix, deviceSumMatrix, threadsMatrixSum * sizeof(double), hipMemcpyDeviceToHost);

  // Check for errors (all values should be height, width, or width * height, respectively)
  float maxErrorColumns = 0.0f;
  float maxErrorRows = 0.0f;
  float maxErrorMatrix = 0.0f;

  for (int i = 0; i < width; i++)
  {
    maxErrorColumns = fmax(maxErrorColumns, fabs(hostSumColumns[i] - height));
  }

  for (int i = 0; i < height; i++)
  {
    maxErrorRows = fmax(maxErrorRows, fabs(hostSumRows[i] - width));
  }

  for (int i = 0; i < threadsMatrixSum; i++)
  {
    maxErrorMatrix = fmax(maxErrorMatrix, fabs(hostSumMatrix[i] - width * height));
  }

  std::cout << "Width: " << width << ", True Width (Pitch / Bytes): " << (float) (pitch) / sizeof(float) << std::endl;
  std::cout << "Max error for columns sum: " << maxErrorColumns << std::endl;
  std::cout << "Max error for rows sum: " << maxErrorRows << std::endl;
  std::cout << "Max error for special matrix sum: " << maxErrorMatrix << std::endl;

  // Free device memory
  hipFree(deviceMatrix);
  hipFree(deviceSumColumns);
  hipFree(deviceSumRows);
  hipFree(deviceSumMatrix);

  // Free host memory
  free(hostMatrix);
  free(hostSumColumns);
  free(hostSumRows);
  free(hostSumMatrix);

  return 0;
}