
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
  {
    y[i] = x[i] + y[i];
  }
}

int main(void)
{
  // Arrays of 16M elements
  int N = 1 << 24;

  // Allocate Unified Memory -- accessible from CPU or GPU (https://developer.nvidia.com/blog/unified-memory-in-cuda-6/)
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 16M elements on the GPU, this function launches one GPU thread to run add
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;

  for (int i = 0; i < N; i++)
  {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }

  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}