
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;

  // Each thread has to take care of N / blockDim sums
  for (int i = index; i < n; i += stride)
  {
    y[i] = x[i] + y[i];
  }
}

int main(void)
{
  // Arrays of 16M elements
  int N = 1 << 24;

  // Allocate Unified Memory, accessible from CPU or GPU (https://developer.nvidia.com/blog/unified-memory-in-cuda-6/)
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on the GPU
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;

  for (int i = 0; i < N; i++)
  {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  }

  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}